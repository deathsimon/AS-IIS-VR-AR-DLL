#include "hip/hip_runtime.h"
#include "stdio.h"
#include "RoomFusionDLL.cuh"

// ��l��CUDA�A�����o�Ө�Ƥ]�i�H���I�s�A�ϥ�CUDA���w�]�ȴNOK
bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);
	fprintf(stderr, "CUDA Ready!\n");



	return true;
}

// Wrapper��ơA�I�s����GPU�������`�׮M��
void runGPUApplyDepth(unsigned char* image, float* depth, int imageWidth, int imageHeight, float threshold){
	dim3 dimBlock(imageHeight, 1);
	dim3 dimGrid(imageWidth, 1);
	gpuApplyDepth << <dimGrid, dimBlock >> >(image, depth, imageWidth, imageHeight, threshold);
}
// Wrapper��ơA�I�s����GPU�������`�׮ե�(�|��έץ�)
void runGPUApplyCorrection(float* depth, int imageWidth, int imageHeight,
	float left_slope, float left_inter, float left_p1x, float left_p1y, float left_p2x, float left_p2y,
	float right_slope, float right_inter, float right_p1x, float right_p1y, float right_p2x, float right_p2y,
	float top_slope, float top_inter, float top_p1x, float top_p1y, float top_p2x, float top_p2y,
	float down_slope, float down_inter, float down_p1x, float down_p1y, float down_p2x, float down_p2y
	){
	dim3 dimBlock(imageHeight, 1);
	dim3 dimGrid(imageWidth, 1);
	gpuApplyCorrection << <dimGrid, dimBlock >> >(depth, imageWidth, imageHeight, 
		left_slope, left_inter, left_p1x, left_p1y, left_p2x, left_p2y,
		right_slope, right_inter, right_p1x, right_p1y, right_p2x, right_p2y,
		top_slope, top_inter, top_p1x, top_p1y, top_p2x, top_p2y,
		down_slope, down_inter, down_p1x, down_p1y, down_p2x, down_p2y
		);
}
// �bGPU�W����`�׮M�ΡA������l�v��(image�A4-channel)�P�`�׸��(depth�A1-channel)�A��W�Lthreshold�����������ܦ��z��
__global__  void gpuApplyDepth(unsigned char* image, float* depth, int imageWidth, int imageHeight, float threshold){
	int w = blockIdx.x;
	int h = threadIdx.x;
	if (w < imageWidth && h < imageHeight){ // ���T�O�S���W�X�d��
		int positionIndex = h * imageWidth + w; // �ѩ�image�O�@���}�C�A�ݭn�ۦ�p��Ypixel����m
		int pixelIndex = positionIndex * 4;  // �]���O4-channel�A�]���@���n��4��byte�~�O�@��pixel
		float depthVal = depth[positionIndex]; // ���X�`�׭�
		if (depthVal > threshold){
			// �����ܦ��z��
			image[pixelIndex + 0] = 0;
			image[pixelIndex + 1] = 128; // �o�Ӻ��ϰ�O���եΪ��A�bUnity���O�ݤ��쪺
			image[pixelIndex + 2] = 0;
			image[pixelIndex + 3] = 0;
		}
	}
}
// �bGPU�W�i��`�׭ץ��A����b�|��Τ���
__global__ void gpuApplyCorrection(float* depth, int imageWidth, int imageHeight,
	float left_slope, float left_inter, float left_p1x, float left_p1y, float left_p2x, float left_p2y,
	float right_slope, float right_inter, float right_p1x, float right_p1y, float right_p2x, float right_p2y,
	float top_slope, float top_inter, float top_p1x, float top_p1y, float top_p2x, float top_p2y,
	float down_slope, float down_inter, float down_p1x, float down_p1y, float down_p2x, float down_p2y
	)
{
	int w = blockIdx.x;
	int h = threadIdx.x;
	if (w < imageWidth && h < imageHeight){
		int positionIndex = h * imageWidth + w;
		int y = imageHeight - h - 1; // Y�y�млݭn�A�ˡA�H����Y�b�ѤU�ӤW���W�h
		if (
				// ���O�bGPU�W�M�Υ|���䪺�ץ�
				gpuIsRightSide(w, y, left_slope, left_inter, left_p1x, left_p1y, left_p2x, left_p2y) &&
				gpuIsLeftSide(w, y, right_slope, right_inter, right_p1x, right_p1y, right_p2x, right_p2y) &&
				gpuIsDownSide(w, y, top_slope, top_inter, top_p1x, top_p1y, top_p2x, top_p2y) &&
				gpuIsUpSide(w, y, down_slope, down_inter, down_p1x, down_p1y, down_p2x, down_p2y)
			)
		{
			// keep depth
		}
		else{
			// no depth
			depth[positionIndex] = 0.0f;
		}
	}
}

// �H�U�|�Ө�ƬO��GPU�ˬd�Y�@���I�O���O�b�Y�@���u���k��/�W��/����/�U��
// �ѼơGpx�Bpy�G�n�ˬd���I
//		 slope�ByIntercept�G�ӱ��u���ײv�PY�I�Z
//       p1x, p1y, p2x, p2y�G�ӱ��u�g�L������I(p1�Pp2)���y��

__device__ bool gpuIsRightSide(float px, float py, float slope, float yIntercept, float p1x, float p1y, float p2x, float p2y){
	if (nearlyEqual(p2y, p1y)){ // horz
		return false;
	}
	else if (nearlyEqual(p2x, p1x)){ // vertical
		return px > p1x;
	}
	float cSolution = (slope*px) + yIntercept;
	if (py > cSolution){
		return p2x <= p1x;
	}
	else{
		return p2x > p1x;
	}
}
__device__ bool gpuIsUpSide(float px, float py, float slope, float yIntercept, float p1x, float p1y, float p2x, float p2y){
	if (nearlyEqual(p2x - p1x, 0)){ // vertical
		return false;
	}
	if (slope > 0){
		return gpuIsLeftSide(px, py, slope, yIntercept, p1x, p1y, p2x, p2y);
	}
	else{
		return gpuIsRightSide(px, py, slope, yIntercept, p1x, p1y, p2x, p2y);
	}
}
__device__ bool gpuIsLeftSide(float px, float py, float slope, float yIntercept, float p1x, float p1y, float p2x, float p2y){
	return !gpuIsRightSide(px, py, slope, yIntercept, p1x, p1y, p2x, p2y);
}
__device__ bool gpuIsDownSide(float px, float py, float slope, float yIntercept, float p1x, float p1y, float p2x, float p2y){
	return !gpuIsUpSide(px, py, slope, yIntercept, p1x, p1y, p2x, p2y);
}